#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}

__global__ void subtract(int *a, int *b, int *c){
	*c = *a - *b;
}

__global__ void multiply(int *a, int *b, int *c){
	*c = *a * *b;
}

__global__ void divide(int *a, int *b, int *c){
	*c = *a / *b;
}

int main(void){
	int a, b, c; 			//host copies of a, b, c
	int *d_a, *d_b, *d_c;	//device copies of a, b, c
	int size = sizeof(int);
	
	//Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	
	//Setup input values
	a = 2;
	b = 7;
	
	//Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	//Launch add() kernel on GPU
	add<<<1,1>>>(d_a,d_b,d_c);
	
	//Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("%d + %d = %d\n",a,b,c);
	
	//Launch subtract() kernel on GPU
	subtract<<<1,1>>>(d_a,d_b,d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("%d - %d = %d\n",a,b,c);
	
	//Launch multiply() kernel on GPU
	multiply<<<1,1>>>(d_a,d_b,d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("%d * %d = %d\n",a,b,c);
	
	//Launch divide() kernel on GPU
	divide<<<1,1>>>(d_a,d_b,d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("%d / %d = %d\n",a,b,c);
	
	//Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return 0;
}